
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
using namespace std;
const int X = 1024;
size_t threadsPerBlock;
size_t numberOfBlocks;

void A_reset(float *A)
{
    for (int i = 0; i < X; i++)
    {
        for (int j = 0; j < i; j++)
            A[i*X+j] = 0;
        A[i*X+i] = 1.0;
        for (int j = i + 1; j < X; j++)
            A[i*X+j] = rand();
    }
    for (int k = 0; k < X; k++)
        for (int i = k + 1; i < X; i++)
            for (int j = 0; j < X; j++)
                A[i*X+j] += A[k*X+j];
}

// 串行:
void normal(int n, float *A)
{
    for (int k = 0; k < n; k++)
    {
        for (int j = k + 1; j < n; j++)
        {
            A[k*n+j] /= A[k*n+k];
        }
        A[k*n+k] = 1.0;
        for (int i = k + 1; i < n; i++)
        {
            for (int j = k + 1; j < n; j++)
            {
                A[i*n+j] -= A[i*n+k] * A[k*n+j];
            }
            A[i*n+k] = 0;
        }
    }
}

//除法:
__global__ void division_kernel(float *A, int k, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index+k+1; i < n ; i += stride)
    {
        float element = A[k*n+k];
        float temp = A[k*n+i];
        A[k*n+i] = (float)temp / element;
    }
    return;
}

//消元:
__global__ void eliminate_kernel(float *A, int k, int N)
{
    int tx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tx == 0)
        A[k*N+k] = 1;  //对角线元素设为 1
    int row = k + 1 + blockIdx.x; //每个块负责一行
    while (row < N)
    {
        int tid = threadIdx.x;
        while (k + 1 + tid < N)
        {
            int col = k + 1 + tid;
            float temp_1 = A[row*N+col];
            float temp_2 = A[row*N+k];
            float temp_3 = A[k*N+col];
            A[row*N+col] = temp_1 - temp_2 * temp_3;
            tid = tid + blockDim.x;
        }
        __syncthreads(); //块内同步
        if (threadIdx.x == 0)
        {
            A[row*N+k] = 0;
        }
        row += gridDim.x;
    }
    return;
}

//并行算法
void sp(int n, float *A)
{
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    threadsPerBlock = 32;
    numberOfBlocks = 32 * numberOfSMs;

    for (int k = 0; k < n; k++)
    {
        division_kernel<<<numberOfBlocks, threadsPerBlock>>>(A, k, n); 
        hipDeviceSynchronize();  
        eliminate_kernel<<<numberOfBlocks, threadsPerBlock>>>(A, k, n); 
        hipDeviceSynchronize();
    }

}
int main()
{
    
    size_t size = X * X * sizeof(float);
    float *A;
    hipMallocManaged(&A, size);
    
    int step = 64;
    clock_t start ,finish_1,finish_2;
    for (int i = step; i <= X; i += step)
    {
        //串行
        A_reset(A);
        start = clock();
        normal(i, A);
        finish_1 = clock();
        float time_1 = ( finish_1 - start)/float (CLOCKS_PER_SEC);
        //并行
        A_reset(A);
        start = clock();
        sp(i, A);
        finish_2 = clock();
        float time_2 = ( finish_2 - start)/float (CLOCKS_PER_SEC);
        cout<<fixed << setprecision(6);
        cout<< time_1 << "    " << time_2 << endl;
        
    }
    hipFree(A);
    cout << "hello" << endl;
    return 0;
}